#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define ID3_BASE_WIDTH		3	
#define ID3_BASE_HEIGHT		6
#define ID3_MID_WIDTH		1
#define ID3_THRESHOLD		.19f	//definitely needs to be changed
#define ID3_SKIP_AMOUNT		1 		//amount to skip in pixels, we can change this to be multiplied by scale if necessary/desirable


//This identifier is 3 vertical bars going dark light dark
__global__ 
void ID3kernel(float* intImage, size_t stride, int* offsets, int windowSize, int numSubWindows, int scale, int* faceDetected, float* results, float* heatMap) {

	int threadNum = blockIdx.x * blockDim.x + threadIdx.x;
	if(threadNum < numSubWindows){
		int startX = offsets[threadNum]/(stride);
		int startY = offsets[threadNum]%stride;
		float maxFitValue = 0.0f;
		
		for (int i = startX; (i+ID3_BASE_WIDTH*scale) < (startX+windowSize); i = i+ID3_SKIP_AMOUNT){ //use ID3_SKIP_AMOUNT * scale for it to scale up as identifier scales
			for (int j = startY; (j+ID3_BASE_HEIGHT*scale) < (startY + windowSize); j = j+ID3_SKIP_AMOUNT){
				// take important corners from image
				float upperLeft 		= intImage[i*stride + j];
				float upperRight 		= intImage[(i+ID3_BASE_WIDTH*scale)*stride + j];
				
				float midLeftTop 		= intImage[(i+ID3_BASE_WIDTH*scale/2 - ID3_MID_WIDTH*scale/2)*stride + j];
				float midRightTop		= intImage[(i+ID3_BASE_WIDTH*scale/2 + ID3_MID_WIDTH*scale/2)*stride + j];
				float midLeftBot 		= intImage[(i+ID3_BASE_WIDTH*scale/2 - ID3_MID_WIDTH*scale/2)*stride + j+ID3_BASE_HEIGHT*scale];
				float midRightBot		= intImage[(i+ID3_BASE_WIDTH*scale/2 + ID3_MID_WIDTH*scale/2)*stride + j+ID3_BASE_HEIGHT*scale];
				
				float lowerLeft 		= intImage[i*stride + j+(ID3_BASE_HEIGHT*scale)];
				float lowerRight 		= intImage[(i+ID3_BASE_WIDTH*scale)*stride + j+(ID3_BASE_HEIGHT*scale)];
			
				//calculate fit value based on identifier (hard-coded)
				// float fitValue = (midRightBot + midLeftTop - midRightTop - midLeftBot)*2.0 - lowerRight - upperLeft + upperRight + lowerLeft;
				float fitValue = 2.0*(midRightBot - midLeftBot - midRightTop + midLeftTop) - (lowerRight - lowerLeft - upperRight + upperLeft) ;
				
				if(fitValue < 0)	
					fitValue = -fitValue;
				
				if(fitValue > maxFitValue){
					maxFitValue = fitValue;
				}
			}
		}
		float goodnessValue = maxFitValue/(ID3_BASE_WIDTH*scale*ID3_BASE_HEIGHT*scale); // goodnessValue = fit/area
	
		results[threadNum] = goodnessValue;
		
		if(goodnessValue > ID3_THRESHOLD){
			faceDetected[threadNum] = 1;

			// for(int i = 0; i < windowSize; ++i){
			// 	for(int j = 0; j < windowSize; ++j){
			// 		heatMap[offsets[threadNum] + i*stride + j] = heatMap[offsets[threadNum] + i*stride + j] + 1.0f;
			// 	}
			// }
		}
	}
}
