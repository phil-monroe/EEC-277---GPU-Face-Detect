#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include "integral.h"
#include "helpers.cu"

#define THREADS_PER_BLOCK 32




__global__ 
void horizontal_kernel(float* data, int rows, int cols, size_t stride ) {
    // start from row 0
	int row = blockIdx.x * blockDim.x + threadIdx.x;

	if(row < rows){
		for(int col = 1; col<cols; ++col){
				data[row*stride + col] = data[row*stride + col] + data[row*stride + col-1];
		}
	}

	
}

__global__ 
void vertical_kernel(float* data, int rows, int cols, size_t stride ) {
    // Start from column 1
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(col < cols){
		for(int row = 1; row<rows; ++row){
			data[row*stride + col] = data[row*stride + col] + data[(row-1)*stride + col] ;
		}
	}
}



void cuda_integrate_image(float* data, int rows, int cols, size_t stride){
	float *dev_data, *dev_fin_data;
	hipMalloc( &dev_data, rows*cols*sizeof(float));
	hipMalloc( &dev_fin_data, rows*cols*sizeof(float));
	
	checkCUDAError("malloc");
	
	hipMemcpy(dev_data, data, rows*cols*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_fin_data, data, rows*cols*sizeof(float), hipMemcpyHostToDevice);
	
	checkCUDAError("memcpy host to device");
	
	int num_blocks = rows < THREADS_PER_BLOCK ? 1 : rows/THREADS_PER_BLOCK + 1;
	
	horizontal_kernel<<<num_blocks , THREADS_PER_BLOCK>>>(dev_data, rows, cols, stride);
	
	num_blocks = cols < THREADS_PER_BLOCK ? 1 : cols/THREADS_PER_BLOCK + 1;
	
	hipDeviceSynchronize();
	checkCUDAError("horizontal kernel");
	
	vertical_kernel<<<num_blocks , THREADS_PER_BLOCK>>>(dev_data, rows, cols, stride);
	hipDeviceSynchronize();
	checkCUDAError("vertical kernel");
	
	hipMemcpy(data, dev_data, rows*cols*sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAError("memcpy device to host");
	
	hipFree(dev_data);
	hipFree(dev_fin_data);
	
	checkCUDAError("free");
	
}