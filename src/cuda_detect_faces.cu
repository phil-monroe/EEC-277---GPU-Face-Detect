#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include "cuda_detect_faces.h"
#include "cuda_helpers.h"
#include "cudpp.h"
#include "identify1.cu"

#define TH_PER_BLOCK 64


void cuda_detect_faces(float* intImg, int rows, int cols, size_t stride, int* windowOffsets, int numWindows, int windowSize){
	CUDPPResult res;
	
	float* results = (float*) malloc(numWindows*sizeof(float));
	float* results2 = (float*) malloc(numWindows*sizeof(float));
	float* results_d;
	float* intImg_d;
	int*		winOffsets_d;
	hipMalloc(&results_d, numWindows*sizeof(float));
	hipMalloc(&intImg_d, rows*cols*sizeof(float));
	hipMalloc(&winOffsets_d, numWindows*sizeof(int));
	checkCUDAError("malloc");
	
	float* results_d2;
	float* intImg_d2;
	int*		winOffsets_d2;
	hipMalloc(&results_d2, numWindows*sizeof(float));
	hipMalloc(&winOffsets_d2, numWindows*sizeof(int));
	checkCUDAError("malloc2");
	
	
	hipMemcpy(winOffsets_d, windowOffsets, numWindows*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(intImg_d, intImg, rows*cols*sizeof(float), hipMemcpyHostToDevice);
	checkCUDAError("memcpy");
	
	hipMemcpy(winOffsets_d2, windowOffsets, numWindows*sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError("memcpy2");
	
	ID1kernel<<<1, TH_PER_BLOCK>>>(winOffsets_d, windowSize, windowSize/10, intImg_d, stride, numWindows, results_d);
	ID1kernel<<<1, TH_PER_BLOCK>>>(winOffsets_d2, windowSize, windowSize/15, intImg_d, stride, numWindows, results_d2);
	hipDeviceSynchronize();
	checkCUDAError("kernel");

	hipMemcpy(results, results_d, numWindows*sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAError("results memcpy");
	
	hipMemcpy(results2, results_d2, numWindows*sizeof(float), hipMemcpyDeviceToHost);
	checkCUDAError("results memcpy2");


	for(size_t i = 0; i < numWindows; ++i){
		if(results[i] > 1.0f || results[i] < -1.0f){
			intImg[windowOffsets[i]] = 1.0;
		}
		printf("%d - %f - %d\n", i, results[i], windowOffsets[i]);
	}
	
	for(size_t i = 0; i < numWindows; ++i){
		if(results[i] > 1.0f || results[i] < -1.0f){
			intImg[windowOffsets[i]] = 1.0;
		}
		printf("%d - %f - %d\n", i, results2[i], windowOffsets[i]);
	}
	
	// printf("run 2:\n");
	
	// float* results_d2;
	// 	float* intImg_d2;
	// 	int*		winOffsets_d2;
	// 	hipMalloc(&results_d2, numWindows*sizeof(float));
	// 	hipMalloc(&winOffsets_d2, numWindows*sizeof(int));
	// 	checkCUDAError("malloc");
	// 	
	// 	
	// 	hipMemcpy(winOffsets_d2, windowOffsets, numWindows*sizeof(int), hipMemcpyHostToDevice);
	// 	checkCUDAError("memcpy");
	// 	
	// 	ID1kernel<<<1, TH_PER_BLOCK>>>(winOffsets_d2, windowSize, windowSize/15, intImg_d, stride, numWindows, results_d2);
	// 	hipDeviceSynchronize();
	// 	checkCUDAError("kernel");
	// 
	// 	hipMemcpy(results, results_d2, numWindows*sizeof(float), hipMemcpyDeviceToHost);
	// 	checkCUDAError("results memcpy");
	// 
	// 
	// 	for(size_t i = 0; i < numWindows; ++i){
	// 		if(results[i] > 1.0f || results[i] < -1.0f){
	// 			intImg[windowOffsets[i]] = 1.0;
	// 		}
	// 		printf("%d - %f - %d\n", i, results[i], windowOffsets[i]);
	// 	}

}