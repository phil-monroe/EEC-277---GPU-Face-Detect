#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

// checkCUDAError -------------------------------------------------------------
//		Convience method to check for cuda errors.
//		@param msg - Unique identifier to help debug.
//
//		From Dr Dobbs "CUDA: Supercomputing for the masses, Part 3"
//		http://drdobbs.com/architecture-and-design/207200659      
//-----------------------------------------------------------------------------
void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}
