#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "identify_glasses.h"

#define FRAME_HEIGHT = 2
#define BASE_WIDTH = 4
#define BASE_HEIGHT = 4
#define THRESHOLD = 0.85 //definitely needs to be changed
#define SKIP_AMOUNT = 4 //amount to skip in pixels, we can change this to be multiplied by scale if necessary/desirable

//This identifier is the glasses identifier with 3 horizontal bars going:
//light 
//dark 
//light
__global__ 
void ID4kernel(int* xVals, int* yVals, int windowSize, int scale, float* intImage, size_t stride, bool* results ) {

	int threadNum = blockIdx.x * blockDim.x + threadIdx.x;
	int startX = xVals[threadNum];
	int startY = yVals[threadNum];
	for (int i = startX; (i+BASE_WIDTH*scale) < (startX+windowSize); i = i+SKIP_AMOUNT){ //use SKIP_AMOUNT * scale for it to scale up as identifier scales
		for (int j = startY; (j+(BASE_HEIGHT)*scale) < (startY + windowSize); j = j+SKIP_AMOUNT){
			// take important corners from image
			int upperLeft 		= intImage[i*stride + j];
			int upperRight 		= intImage[(i+BASE_WIDTH*scale)*stride + j];
			
			int midLeftTop 		= intImage[i*stride + j + ((BASE_HEIGHT>>1 - FRAME_HEIGHT>>1) * scale)];
			int midRightTop 		= intImage[(i+BASE_WIDTH*scale)*stride + j + ((BASE_HEIGHT>>1 - FRAME_HEIGHT>>1) * scale)];
			
			int midLeftBot 		= intImage[i*stride + j + ((BASE_HEIGHT>>1 + FRAME_HEIGHT>>1) * scale)];
			int midRightBot 		= intImage[(i+BASE_WIDTH*scale)*stride + j + ((BASE_HEIGHT>>1 + FRAME_HEIGHT>>1) * scale)];
			
			int lowerLeft 		= intImage[i*stride + j+((FRAME_HEIGHT+EYE_HEIGHT)*scale)];
			int lowerRight 		= intImage[(i+BASE_WIDTH*scale)*stride + j+((FRAME_HEIGHT+EYE_HEIGHT)*scale)];
			
			//calculate fit value based on identifier (hard-coded)
			int fitValue = upperLeft - lowerLeft - upperRight + lowerRight + (midRightTop + midRightBot - midLeftTop - midRightBot)<<1;
			float goodnessValue = fitValue*1.0f/(BASE_WIDTH*scale*(FRAME_HEIGHT + EYE_HEIGHT)*scale); // goodnessValue = fit/area
			
			results[i*stride + j] = (goodnessValue>THRESHOLD);
		}
	}
    
}
