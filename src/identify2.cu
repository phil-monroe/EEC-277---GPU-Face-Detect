#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define ID2_BASE_WIDTH			8
#define ID2_BASE_HEIGHT			4
#define ID2_THRESHOLD			100.0f	//definitely needs to be changed
#define ID2_SKIP_AMOUNT			4 			//amount to skip in pixels, we can change this to be multiplied by scale if necessary/desirable

//This identifier is 2 horizontal bars with dark (negative) on top and light (positive) on bottom
__global__ 
void ID2kernel(float* intImage, size_t stride, int* offsets, int windowSize, int numSubWindows, int scale, int* faceDetected, float* results, float* heatMap) {
	int threadNum = blockIdx.x * blockDim.x + threadIdx.x;
	if(threadNum < numSubWindows){
		float maxFitValue = 0.0f;
		int startX = offsets[threadNum]/(stride);
		int startY = offsets[threadNum]%stride;
		for (int i = startX; (i+ID2_BASE_WIDTH*scale) < (startX+windowSize); i = i+ID2_SKIP_AMOUNT){ //use ID2_SKIP_AMOUNT * scale for it to scale up as identifier scales
			for (int j = startY; (j+ID2_BASE_HEIGHT*scale) < (startY + windowSize); j = j+ID2_SKIP_AMOUNT){
				// take important corners from image
				float upperLeft 		= intImage[i*stride + j];
				float upperRight 		= intImage[(i+ID2_BASE_WIDTH*scale)*stride + j];
				float midLeft 			= intImage[i*stride + j+(ID2_BASE_HEIGHT*scale>>1)];
				float midRight 		= intImage[(i+ID2_BASE_WIDTH*scale)*stride + j+(ID2_BASE_HEIGHT*scale>>1)];
				float lowerLeft 		= intImage[i*stride + j+(ID2_BASE_HEIGHT*scale)];
				float lowerRight 		= intImage[(i+ID2_BASE_WIDTH*scale)*stride + j+(ID2_BASE_HEIGHT*scale)];

				//calulate fit value based on identifier (hard-coded)
				float fitValue = midLeft*2 - midRight*2 - upperLeft + lowerRight + upperRight - lowerLeft;

				if(fitValue > maxFitValue){
					maxFitValue = fitValue;
				}
			}
		}
		float goodnessValue = maxFitValue;//(ID2_BASE_WIDTH*scale*ID2_BASE_HEIGHT*scale); // goodnessValue = fit/area
		results[threadNum] = goodnessValue;

		if(goodnessValue > ID2_THRESHOLD){
			faceDetected[threadNum] = 1;

			for(int i = 0; i < windowSize; ++i){
				for(int j = 0; j < windowSize; ++j){
					heatMap[offsets[threadNum] + i*stride + j] = heatMap[offsets[threadNum] + i*stride + j] + 1.0f;
				}
			}
		}
	}
}

