#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "identify1.h"

#define BASE_WIDTH = 8
#define BASE_HEIGHT = 2
#define THRESHOLD = 0.85 //definitely needs to be changed

__global__ 
void ID1kernel(int startX, int startY, int scale, int** integImage ) {
    // take important corners from image
	int upperLeft = integImage[startX][startY];
	int upperRight = integImage[startX+BASE_WIDTH*scale][startY];
	int midLeft = integImage[startX][startY+BASE_HEIGHT*scale];
	int midRight = integImage[startX+BASE_WIDTH*scale][startY+BASE_HEIGHT*scale];
	int lowerLeft = integImage[startX][startY+BASE_HEIGHT*scale<<1];
	int lowerRight = integImage[startX+BASE_WIDTH*scale][startY+BASE_HEIGHT*scale<<1];
	
	//calculate fit value based on identifier (hard-coded)
	int fitValue = midRight<<1-midLeft<<1 + upperLeft - lowerRight - upperRight + lowerLeft;
	float goodnessValue = fitValue*1.0f/(BASE_WIDTH*scale*BASE_HEIGHT*scale<<1); // goodnessValue = fit/area

}
