#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "identify1.h"

#define BASE_WIDTH = 8
#define BASE_HEIGHT = 2
#define THRESHOLD = 0.85 //definitely needs to be changed

__global__ 
void ID1kernel(int startX, int startY, int scale, float* intImage ) {
    // take important corners from image
	int upperLeft 		= intImage[startX][startY];
	int upperRight 	= intImage[startX+BASE_WIDTH*scale][startY];
	int midLeft 		= intImage[startX][startY+BASE_HEIGHT*scale];
	int midRight 		= intImage[startX+BASE_WIDTH*scale][startY+BASE_HEIGHT*scale];
	int lowerLeft 		= intImage[startX][startY+BASE_HEIGHT*scale<<1];
	int lowerRight 	= intImage[startX+BASE_WIDTH*scale][startY+BASE_HEIGHT*scale<<1];
	
	//calculate fit value based on identifier (hard-coded)
	int fitValue = midRight<<1-midLeft<<1 + upperLeft - lowerRight - upperRight + lowerLeft;
	float goodnessValue = fitValue*1.0f/(BASE_WIDTH*scale*BASE_HEIGHT*scale<<1); // goodnessValue = fit/area

}
