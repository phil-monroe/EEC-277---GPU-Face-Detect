#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define BASE_WIDTH	8
#define BASE_HEIGHT	4
#define THRESHOLD		0.85 //definitely needs to be changed
#define SKIP_AMOUNT	4 //amount to skip in pixels, we can change this to be multiplied by scale if necessary/desirable

//This identifier is 2 horizontal bars with light (positive) on top and dark (negative) on bottom
__global__ 
void ID1kernel(int* offsets, int windowSize, int scale, float* intImage, size_t stride, int numSubWindows, float* results) {
	int threadNum = blockIdx.x * blockDim.x + threadIdx.x;
	if(threadNum < numSubWindows){
		float maxFitValue;
		int maxX;
		int maxY;
		int startX = offsets[threadNum]/(stride);
		int startY = offsets[threadNum]%stride;
		for (int i = startX; (i+BASE_WIDTH*scale) < (startX+windowSize); i = i+SKIP_AMOUNT*scale){ //use SKIP_AMOUNT * scale for it to scale up as identifier scales
			for (int j = startY; (j+BASE_HEIGHT*scale) < (startY + windowSize); j = j+SKIP_AMOUNT*scale){
				// take important corners from image
				float upperLeft 		= intImage[i*stride + j];
				float upperRight 		= intImage[(i+BASE_WIDTH*scale)*stride + j];
				float midLeft 			= intImage[i*stride + j+(BASE_HEIGHT*scale>>1)];
				float midRight 		= intImage[(i+BASE_WIDTH*scale)*stride + j+(BASE_HEIGHT*scale>>1)];
				float lowerLeft 		= intImage[i*stride + j+(BASE_HEIGHT*scale)];
				float lowerRight 		= intImage[(i+BASE_WIDTH*scale)*stride + j+(BASE_HEIGHT*scale)];
		
				//calculate fit value based on identifier (hard-coded)
				float fitValue = midRight*2-midLeft*2 + upperLeft - lowerRight - upperRight + lowerLeft;
				maxFitValue = (maxFitValue < fitValue) ? fitValue : maxFitValue;
				maxX = (maxFitValue <= fitValue) ? i : maxX;
				maxY = (maxFitValue <= fitValue) ? j : maxY;
		
			}
		}
		float goodnessValue = maxFitValue/(BASE_WIDTH*scale*BASE_HEIGHT*scale); // goodnessValue = fit/area
		offsets[threadNum] = maxX;//(goodnessValue>THRESHOLD) ? offsets[threadNum] : -1.0;
		results[threadNum] = maxFitValue;//(float) maxX;
	}

    
}

