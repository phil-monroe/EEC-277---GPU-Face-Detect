
#include <hip/hip_runtime.h>
// checkCUDAError -------------------------------------------------------------
//		Convience method to check for cuda errors.
//		@param msg - Unique identifier to help debug.
//
//		From Dr Dobbs "CUDA: Supercomputing for the masses, Part 3"
//		http://drdobbs.com/architecture-and-design/207200659      
//-----------------------------------------------------------------------------
void checkCUDAError(const char *msg) {
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}